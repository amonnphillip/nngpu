#include "hip/hip_runtime.h"
#pragma once
#include <stdio.h>
#include <stdexcept>

#include "convlayer.h"
#include "layersize.h"
#include "hip/hip_runtime.h"
#include "math.h"
#include ""
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void ConvLayer_Forward_cu_test(ConvNode *node, double* filters, LayerSize filterSize, LayerSize layerSize, LayerSize previousLayerSize, double *previousLayerOutput, double *output, int pad)
{
	for (int d2 = 0; d2 < 32; d2++)
	{
		for (int y = 0; y < layerSize.height; y++)
		{
			for (int x = 0; x < layerSize.width; x++)
			{
				int posx = x - pad;
				int posy = y - pad;
				double val = 0;
				double* filter = filters + (filterSize.width * filterSize.height * filterSize.depth * d2);

				for (int filterPosy = 0; filterPosy < filterSize.height; filterPosy++)
				{
					for (int filterPosx = 0; filterPosx < filterSize.width; filterPosx++)
					{
						if (filterPosy + posy >= 0 &&
							filterPosy + posy < previousLayerSize.height &&
							filterPosx + posx >= 0 &&
							filterPosx + posx < previousLayerSize.width)
						{
							for (int d = 0; d < filterSize.depth; d++)
							{
								int index1 = ((filterPosy * filterSize.width) + filterPosx) * filterSize.depth + d;
								int index2 = (((posy + filterPosy) * previousLayerSize.width) + posx + filterPosx) * previousLayerSize.depth + d;

								if (index1 > filterSize.width * filterSize.height * filterSize.depth)
								{
									val = 0;
								}

								if (index2 > previousLayerSize.width * previousLayerSize.height * previousLayerSize.depth)
								{
									val = 0;
								}
								val += filter[index1] * previousLayerOutput[index2];
							}
						}
					}
				}

				//val += node->bias;
				output[((y * layerSize.width) + x) * layerSize.depth + d2] = val;
			}
		}
	}
}

__global__ void ConvLayer_Backward_cu_test(ConvNode *node, double* filters, double* backFilters, LayerSize filterSize, int filterCount, LayerSize layerSize, LayerSize previousLayerSize, LayerSize nextLayerSize, double *previousLayerOutput, double *nextLayerOutput, double *output, int pad, double learnRate)
{
	for (int d2 = 0; d2<filterCount; d2++)
	{
		for (int y = 0; y<layerSize.height; y++)
		{
			for (int x = 0; x<layerSize.width; x++)
			{
				int posx = x - pad;
				int posy = y - pad;
				double* filter = filters + (filterSize.width * filterSize.height * filterSize.depth * d2);
				double* backFilter = backFilters + (filterSize.width * filterSize.height * filterSize.depth * d2);
				double gradient = nextLayerOutput[((layerSize.width * y) + x) * nextLayerSize.depth + d2];

				for (int filterPosy = 0; filterPosy < filterSize.height; filterPosy++)
				{
					for (int filterPosx = 0; filterPosx < filterSize.width; filterPosx++)
					{
						if (filterPosy + posy >= 0 &&
							filterPosy + posy < layerSize.height &&
							filterPosx + posx >= 0 &&
							filterPosx + posx < layerSize.width)
						{
							for (int d = 0; d < filterSize.depth; d++)
							{
								int index1 = ((layerSize.width * (filterPosy + posy)) + filterPosx + posx) * previousLayerSize.depth + d;
								int index2 = ((filterSize.width * filterPosy) + filterPosx) * filterSize.depth + d;

								backFilter[index2] += previousLayerOutput[index1] * gradient;
								output[index1] += filter[index2] * gradient;

								if (index1 == 11)
								{
									printf("gradient x: %i, y: %i nextLayerSize.depth: %i, d2: %i, layerSize.width: %i \n", x, y, nextLayerSize.depth, d2, layerSize.width);
									printf("filterPosx: %i, filterPosy: %i, posx: %i, posy: %i\n", filterPosx, filterPosy, posx, posy);
									printf("gradient %f\n", gradient);
									printf("filter: %i\n", d2);
									printf("d: %i\n", d);
									printf("index1: %i\n index2: %i\n", index1, index2);
								}
							}
						}
					}
				}
			}
		}
	}


	//printf("output[10, 10] %f\n", output[(10 * layerSize.width) + 10]);
	printf("output[1] %f\n", output[11]);

	//node->bias += gradient * learnRate;
}

void ConvLayer_ForwardReference(ConvNode *node, double* filters, LayerSize filterSize, int filterCount, LayerSize layerSize, LayerSize previousLayerSize, double *previousLayerOutput, double *output, int pad)
{
	ConvLayer_Forward_cu_test << <1, 1 >> >(node, filters, filterSize, layerSize, previousLayerSize, previousLayerOutput, output, pad);

	if (hipGetLastError() != hipError_t::hipSuccess)
	{
		throw std::runtime_error("Conv Reference Forward CUDA method returned an error");
	}

	if (hipDeviceSynchronize() != hipError_t::hipSuccess)
	{
		throw std::runtime_error("Conv Reference Forward CUDA syncronize returned an error");
	}
}

void ConvLayer_BackwardReference(ConvNode *node, double* filters, double* backFilters, LayerSize filterSize, int filterCount, LayerSize layerSize, LayerSize previousLayerSize, LayerSize nextLayerSize, double *previousLayerOutput, double *nextLayerOutput, double *output, int pad, double learnRate)
{
	ConvLayer_Backward_cu_test << <1, 1 >> >(node, filters, backFilters, filterSize, filterCount, layerSize, previousLayerSize, nextLayerSize, previousLayerOutput, nextLayerOutput, output, pad, learnRate);

	if (hipGetLastError() != hipError_t::hipSuccess)
	{
		throw std::runtime_error("Conv Reference Forward CUDA method returned an error");
	}

	if (hipDeviceSynchronize() != hipError_t::hipSuccess)
	{
		throw std::runtime_error("Conv Reference Forward CUDA syncronize returned an error");
	}
}