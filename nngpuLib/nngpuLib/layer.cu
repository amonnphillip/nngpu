#include <stdio.h>
#include <stdexcept>

#include <hip/hip_runtime.h>
#include <math.h>

#include <hip/device_functions.h>

#include <hip/hip_runtime_api.h>

void LayerSynchronize()
{
	if (hipGetLastError() != hipError_t::hipSuccess)
	{
		throw std::runtime_error("CUDA method returned an error");
	}

	if (hipDeviceSynchronize() != hipError_t::hipSuccess)
	{
		throw std::runtime_error("CUDA syncronize returned an error");
	}
}