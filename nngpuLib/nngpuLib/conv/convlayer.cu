#include "hip/hip_runtime.h"
#pragma once

#include "convlayer.h"
#include "layersize.h"
#include "hip/hip_runtime.h"
#include "math.h"
#include ""
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdexcept>

__global__ void ConvLayer_Forward_cu(ConvNode *node, double* filters, LayerSize filterSize, LayerSize layerSize, LayerSize previousLayerSize, double *previousLayerOutput, double *output, int pad)
{
	int posx = blockIdx.x - pad;
	int posy = blockIdx.y - pad;
	double val = 0;
	double* filter = filters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.z);

	for (int filterPosy = 0; filterPosy < filterSize.height;filterPosy++)
	{
		for (int filterPosx = 0; filterPosx < filterSize.width; filterPosx++)
		{
			if (filterPosy + posy >= 0 &&
				filterPosy + posy < previousLayerSize.height &&
				filterPosx + posx >= 0 &&
				filterPosx + posx < previousLayerSize.width)
			{
				for (int d = 0; d < filterSize.depth; d++)
				{
					int index1 = ((filterPosy * filterSize.width) + filterPosx) * filterSize.depth + d;
					int index2 = (((posy + filterPosy) * previousLayerSize.width) + posx + filterPosx) * previousLayerSize.depth + d;

					if (index1 > filterSize.width * filterSize.height * filterSize.depth)
					{
						val = 0;
					}

					if (index2 > previousLayerSize.width * previousLayerSize.height * previousLayerSize.depth)
					{
						val = 0;
					}
					val += filter[index1] * previousLayerOutput[index2];
				}
			}
		}
	}

	//val += node->bias;
	output[((blockIdx.y * layerSize.width) + blockIdx.x) * layerSize.depth + blockIdx.z] = val;
}


__global__ void ConvLayer_Backward_cu(ConvNode *node, double* filters, double* backFilters, LayerSize filterSize, int filterCount, LayerSize layerSize, LayerSize previousLayerSize, LayerSize nextLayerSize, double *previousLayerOutput, double *nextLayerOutput, double *output, int pad, double learnRate)
{
	int posx = blockIdx.x - pad;
	int posy = blockIdx.y - pad;
	double* filter = filters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.z);
	double* backFilter = backFilters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.z);
	double gradient = nextLayerOutput[((layerSize.width * blockIdx.y) + blockIdx.x) * nextLayerSize.depth + blockIdx.z];

	for (int filterPosy = 0; filterPosy < filterSize.height; filterPosy++)
	{
		for (int filterPosx = 0; filterPosx < filterSize.width; filterPosx++)
		{
			if (filterPosy + posy >= 0 &&
				filterPosy + posy < layerSize.height &&
				filterPosx + posx >= 0 &&
				filterPosx + posx < layerSize.width)
			{
				for (int d = 0; d < filterSize.depth; d++)
				{
					int index1 = ((layerSize.width * (filterPosy + posy)) + filterPosx + posx) * previousLayerSize.depth + d;
					int index2 = ((filterSize.width * filterPosy) + filterPosx) * filterSize.depth + d;

					backFilter[index2] += previousLayerOutput[index1] * gradient;
					output[index1] += filter[index2] * gradient;
				}
			}
		}
	}

	//node->bias += gradient * learnRate;
}

__global__ void ConvLayer_Backward_update_back_filters_cu(ConvNode *node, double* filters, double* backFilterCollation, double* backFilters, LayerSize filterSize, int filterCount, LayerSize layerSize, LayerSize previousLayerSize, LayerSize nextLayerSize, double *previousLayerOutput, double *nextLayerOutput, double *output, int pad, double learnRate)
{
	/*
	blockIdx.xxx is layer size

	int posx = blockIdx.x - pad;
	int posy = blockIdx.y - pad;
	double* filter = filters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.z);
	double* backFilter = backFilters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.z);
	double gradient = nextLayerOutput[((layerSize.width * blockIdx.y) + blockIdx.x) * nextLayerSize.depth + blockIdx.z];

	for (int filterPosy = 0; filterPosy < filterSize.height; filterPosy++)
	{
		for (int filterPosx = 0; filterPosx < filterSize.width; filterPosx++)
		{
			if (filterPosy + posy >= 0 &&
				filterPosy + posy < layerSize.height &&
				filterPosx + posx >= 0 &&
				filterPosx + posx < layerSize.width)
			{
				for (int d = 0; d < filterSize.depth; d++)
				{
					int index1 = ((layerSize.width * (filterPosy + posy)) + filterPosx + posx) * previousLayerSize.depth + d;
					int index2 = ((filterSize.width * filterPosy) + filterPosx) * filterSize.depth + d;

					backFilter[index2] += previousLayerOutput[index1] * gradient;
					output[index1] += filter[index2] * gradient;
				}
			}
		}
	}*/

	int posx = blockIdx.x - pad;
	int posy = blockIdx.y - pad;
	//double* filter = filters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.z);
	unsigned int collationFilterGroupSize = filterSize.width * filterSize.height * filterSize.depth * layerSize.width * layerSize.height;
	double* backFilter = backFilterCollation + (collationFilterGroupSize * blockIdx.z);// *blockIdx.x * blockIdx.y);
	backFilter += (filterSize.width * filterSize.height * filterSize.depth * layerSize.width * blockIdx.y);
	backFilter += (filterSize.width * filterSize.height * filterSize.depth * blockIdx.x);
	double gradient = nextLayerOutput[((layerSize.width * blockIdx.y) + blockIdx.x) * nextLayerSize.depth + blockIdx.z];

	for (int filterPosy = 0; filterPosy < filterSize.height; filterPosy++)
	{
		for (int filterPosx = 0; filterPosx < filterSize.width; filterPosx++)
		{
			if (filterPosy + posy >= 0 &&
				filterPosy + posy < layerSize.height &&
				filterPosx + posx >= 0 &&
				filterPosx + posx < layerSize.width)
			{
				for (int d = 0; d < filterSize.depth; d++)
				{
					int index1 = ((layerSize.width * (filterPosy + posy)) + filterPosx + posx) * previousLayerSize.depth + d;
					int index2 = ((filterSize.width * filterPosy) + filterPosx) * filterSize.depth + d;

					backFilter[index2] += previousLayerOutput[index1] * gradient;
					//output[index1] += filter[index2] * gradient;
				}
			}
		}

	}
	/*



	for (int d2 = 0; d2<filterCount; d2++)
	{
		//double* filter = filters + (filterSize.width * filterSize.height * filterSize.depth * d2);
		double* backFilter = backFilters + (filterSize.width * filterSize.height * filterSize.depth * d2);

		for (int y = 0; y<layerSize.height; y++)
		{
			for (int x = 0; x<layerSize.width; x++)
			{
				int posx = x - pad;
				int posy = y - pad;
				double gradient = nextLayerOutput[((layerSize.width * y) + x) * nextLayerSize.depth + d2];

				for (int filterPosy = 0; filterPosy < filterSize.height; filterPosy++)
				{
					for (int filterPosx = 0; filterPosx < filterSize.width; filterPosx++)
					{
						if (filterPosy + posy >= 0 &&
							filterPosy + posy < layerSize.height &&
							filterPosx + posx >= 0 &&
							filterPosx + posx < layerSize.width)
						{
							for (int d = 0; d < filterSize.depth; d++)
							{
								int index1 = ((layerSize.width * (filterPosy + posy)) + filterPosx + posx) * previousLayerSize.depth + d;
								int index2 = ((filterSize.width * filterPosy) + filterPosx) * filterSize.depth + d;

								backFilter[index2] += previousLayerOutput[index1] * gradient;
								//output[index1] += filter[index2] * gradient;
							}
						}
					}
				}
			}
		}
	}

	blockIdx.x = filterPosx
	blockIdx.y = filterPosy
	blockIdx.z = d

	call for each filter = d2

	SHARED MEMORY?
	threadIdx.x = d2 = shared memory?


	if (blockIdx.y + posy >= 0 &&
		blockIdx.y + posy < layerSize.height &&
		blockIdx.x + posx >= 0 &&
		blockIdx.x + posx < layerSize.width)
	{
		int index1 = ((layerSize.width * (blockIdx.y + posy)) + blockIdx.x + posx) * previousLayerSize.depth + blockIdx.z;
		int index2 = ((filterSize.width * blockIdx.y) + blockIdx.x) * filterSize.depth + blockIdx.z;

		backFilter[index2] += previousLayerOutput[index1] * gradient;
	}


	*/



	//blockIdx.z = filter count
	//	blockIdx.x, y = filter x and y
	//	threadIdx.z = depth... not needed?
	//	threadIdx.x = layer x
	//	threadIdx.y = layer y
/*
	double* backFilter = backFilters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.z);
	backFilter += ((filterSize.width * blockIdx.y) + blockIdx.x) * filterSize.depth + threadIdx.z;

	double gradient = nextLayerOutput[((layerSize.width * threadIdx.y) + threadIdx.x) * nextLayerSize.depth + blockIdx.z];
	int posx = threadIdx.x - pad;
	int posy = threadIdx.y - pad;
	if (blockIdx.y + posy >= 0 &&
		blockIdx.y + posy < layerSize.height &&
		blockIdx.x + posx >= 0 &&
		blockIdx.x + posx < layerSize.width)
	{
		for (int d = 0; d < filterSize.depth; d++)
		{
			int index1 = ((layerSize.width * (blockIdx.y + posy)) + blockIdx.x + posx) * previousLayerSize.depth + d;
			*backFilter += previousLayerOutput[index1] * gradient;
		}
	}*/
	/*
	double* backFilter = backFilters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.z);

	for (int y = 0; y < layerSize.height; y++)
	{
		for (int x = 0; x < layerSize.width; x++)
		{
			int posx = x - pad;
			int posy = y - pad;
			double* filter = filters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.z);
			//double* backFilter = backFilters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.z);
			double gradient = nextLayerOutput[((layerSize.width * y) + x) * nextLayerSize.depth + blockIdx.z];

			//for (int filterPosy = 0; filterPosy < filterSize.height; filterPosy++)
			//{
				//for (int filterPosx = 0; filterPosx < filterSize.width; filterPosx++)
				//{
					if (blockIdx.y + posy >= 0 &&
						blockIdx.y + posy < layerSize.height &&
						blockIdx.x + posx >= 0 &&
						blockIdx.x + posx < layerSize.width)
					{
						for (int d = 0; d < filterSize.depth; d++)
						{
							int index1 = ((layerSize.width * (blockIdx.y + posy)) + blockIdx.x + posx) * previousLayerSize.depth + d;
							int index2 = ((filterSize.width * blockIdx.y) + blockIdx.x) * filterSize.depth + d;

							backFilter[index2] += previousLayerOutput[index1] * gradient;
							//output[index1] += filter[index2] * gradient;
						}
					}
				//}
			//}
		}
	}
	*/
	//node->bias += gradient * learnRate;
}

__global__ void ConvLayer_Backward_update_back_filters_collate(double* backFilterCollation, double* backFilters, LayerSize filterSize, LayerSize layerSize)
{
	// do each pixel in each filter!
	// 
    // blockIdx.x = filter x
	// blockIdx.y = filter y
	// blockIdx.z = filter index (count)

	// TODO: I ASSUME THE PAD IS 1!!

	unsigned int collationFilterGroupSize = filterSize.width * filterSize.height * filterSize.depth * layerSize.width * layerSize.height;

	double* backFilter = backFilters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.z) + (blockIdx.y * filterSize.width) + blockIdx.x;
	double* collation = backFilterCollation + (collationFilterGroupSize * blockIdx.z) + (blockIdx.y * filterSize.width) + blockIdx.x;
	for (int y = 0; y < layerSize.height; y++)
	{
		for (int x = 0; x < layerSize.width; x++)
		{
			*backFilter += *collation;
			collation += filterSize.width * filterSize.height * filterSize.depth;
		}
	}
}

__global__ void ConvLayer_Backward_update_output_cu(ConvNode *node, double* filters, double* backFilters, LayerSize filterSize, int filterCount, LayerSize layerSize, LayerSize previousLayerSize, LayerSize nextLayerSize, double *previousLayerOutput, double *nextLayerOutput, double *output, int pad, double learnRate)
{
	/*
	int posx = blockIdx.x - pad;
	int posy = blockIdx.y - pad;
	double* filter = filters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.z);
	double* backFilter = backFilters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.z);
	double gradient = nextLayerOutput[((layerSize.width * blockIdx.y) + blockIdx.x) * nextLayerSize.depth + blockIdx.z];

	for (int filterPosy = 0; filterPosy < filterSize.height; filterPosy++)
	{
		for (int filterPosx = 0; filterPosx < filterSize.width; filterPosx++)
		{
			if (filterPosy + posy >= 0 &&
				filterPosy + posy < layerSize.height &&
				filterPosx + posx >= 0 &&
				filterPosx + posx < layerSize.width)
			{
				for (int d = 0; d < filterSize.depth; d++)
				{
					int index1 = ((layerSize.width * (filterPosy + posy)) + filterPosx + posx) * previousLayerSize.depth + d;
					int index2 = ((filterSize.width * filterPosy) + filterPosx) * filterSize.depth + d;

					backFilter[index2] += previousLayerOutput[index1] * gradient;
					output[index1] += filter[index2] * gradient;
				}
			}
		}
	}*/

	/*
	int posx = blockIdx.x - pad;
	int posy = blockIdx.y - pad;
	double* filter = filters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.z);
	double* backFilter = backFilters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.z);
	double gradient = nextLayerOutput[((layerSize.width * blockIdx.y) + blockIdx.x) * nextLayerSize.depth + blockIdx.z];

	for (int filterPosy = 0; filterPosy < filterSize.height; filterPosy++)
	{
		for (int filterPosx = 0; filterPosx < filterSize.width; filterPosx++)
		{
			if (filterPosy + posy >= 0 &&
				filterPosy + posy < layerSize.height &&
				filterPosx + posx >= 0 &&
				filterPosx + posx < layerSize.width)
			{
				for (int d = 0; d < filterSize.depth; d++)
				{
					int index1 = ((layerSize.width * (filterPosy + posy)) + filterPosx + posx) * previousLayerSize.depth + d;
					int index2 = ((filterSize.width * filterPosy) + filterPosx) * filterSize.depth + d;

					//backFilter[index2] += previousLayerOutput[index1] * gradient;
					output[index1] += filter[index2] * gradient;
				}
			}
		}
	}*/

	// TODO: ASSUMING PAD OF 1!!

	//int posx = blockIdx.x - pad;
	//int posy = blockIdx.y - pad;
	int d = blockIdx.z;

	//unsigned int index1 = ((layerSize.width * (filterPosy + posy)) + filterPosx + posx) * previousLayerSize.depth + d;
	unsigned int index1 = ((layerSize.width * blockIdx.y) + blockIdx.x) * previousLayerSize.depth + d;

	for (int filterIndex = 0; filterIndex < filterCount; filterIndex++)
	{
		double gradient = nextLayerOutput[((layerSize.width * blockIdx.y) + blockIdx.x) * nextLayerSize.depth + filterIndex];

		for (int filterPosy = 0; filterPosy < filterSize.height; filterPosy++)
		{
			for (int filterPosx = 0; filterPosx < filterSize.width; filterPosx++)
			{
				if (blockIdx.x - pad + filterPosx >= 0 &&
					blockIdx.x - pad + filterPosx < layerSize.width &&
					blockIdx.y - pad + filterPosy >= 0 &&
					blockIdx.y - pad + filterPosy < layerSize.height) {
					double* filter = filters + (filterSize.width * filterSize.height * filterSize.depth * filterIndex);
					int index2 = ((filterSize.width * filterPosy) + filterPosx) * filterSize.depth + d;

					output[index1] += filter[index2] * gradient;
				}
			}
		}
	}

	/*

	int posx = ((int)blockIdx.x) - pad;
	int posy = ((int)blockIdx.y) - pad;
	double* filter = filters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.z);
	//double* backFilter = backFilters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.z);
	double gradient = nextLayerOutput[((layerSize.width * blockIdx.y) + blockIdx.x) * nextLayerSize.depth + blockIdx.z];

	//for (int filterPosy = 0; filterPosy < filterSize.height; filterPosy++)
	//{
		//for (int filterPosx = 0; filterPosx < filterSize.width; filterPosx++)
		//{
			if (threadIdx.y + posy >= 0 &&
				threadIdx.y + posy < layerSize.height &&
				threadIdx.x + posx >= 0 &&
				threadIdx.x + posx < layerSize.width)
			{
				for (int d = 0; d < filterSize.depth; d++)
				{
					int index1 = ((layerSize.width * (threadIdx.y + posy)) + threadIdx.x + posx) * previousLayerSize.depth + d;
					int index2 = ((filterSize.width * threadIdx.y) + threadIdx.x) * filterSize.depth + d;

					//backFilter[index2] += previousLayerOutput[index1] * gradient;
					output[index1] += filter[index2] * gradient;

					//__syncthreads();
				}
			}
		//}
	//}*/

	//node->bias += gradient * learnRate;
}

__global__ void ConvLayer_Backward_cu_2(ConvNode *node, double* filters, double* backFilters, LayerSize filterSize, int filterCount, LayerSize layerSize, LayerSize previousLayerSize, LayerSize nextLayerSize, double *previousLayerOutput, double *nextLayerOutput, double *output, int pad, double learnRate)
{
	for (int d2 = 0;d2<filterCount;d2++)
	{
		for (int y = 0; y<layerSize.height; y++)
		{
			for (int x = 0; x<layerSize.width; x++)
			{
				int posx = x - pad;
				int posy = y - pad;
				double* filter = filters + (filterSize.width * filterSize.height * filterSize.depth * d2);
				double* backFilter = backFilters + (filterSize.width * filterSize.height * filterSize.depth * d2);
				double gradient = nextLayerOutput[((layerSize.width * y) + x) * nextLayerSize.depth + d2];

				for (int filterPosy = 0; filterPosy < filterSize.height; filterPosy++)
				{
					for (int filterPosx = 0; filterPosx < filterSize.width; filterPosx++)
					{
						if (filterPosy + posy >= 0 &&
							filterPosy + posy < layerSize.height &&
							filterPosx + posx >= 0 &&
							filterPosx + posx < layerSize.width)
						{
							for (int d = 0; d < filterSize.depth; d++)
							{
								int index1 = ((layerSize.width * (filterPosy + posy)) + filterPosx + posx) * previousLayerSize.depth + d;
								int index2 = ((filterSize.width * filterPosy) + filterPosx) * filterSize.depth + d;

								//backFilter[index2] += previousLayerOutput[index1] * gradient;
								output[index1] += filter[index2] * gradient;
							}
						}
					}
				}
			}
		}
	}



	//node->bias += gradient * learnRate;
}

__global__ void ConvLayer_Update_Backward_filter_cu(double* filters, double* backFilters, LayerSize filterSize, double learnRate)
{
	double* filter = filters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.x);
	double* backFilter = backFilters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.x);

	int size = filterSize.width * filterSize.height * filterSize.depth;
	for (int index = 0; index < size; index++)
	{
		filter[index] += backFilter[index] * learnRate;
	}
}

void ConvLayer_Forward(ConvNode *node, double* filters, LayerSize filterSize, int filterCount, LayerSize layerSize, LayerSize previousLayerSize, double *previousLayerOutput, double *output, int pad)
{
	dim3 blocks(layerSize.width, layerSize.height, filterCount);
	ConvLayer_Forward_cu <<<blocks, 1>>>(node, filters, filterSize, layerSize, previousLayerSize, previousLayerOutput, output, pad);
//	ConvLayer_Forward_cu_2 << <1, 1 >> >(node, filters, filterSize, layerSize, previousLayerSize, previousLayerOutput, output, pad);

	if (hipGetLastError() != hipError_t::hipSuccess)
	{
		throw std::runtime_error("FullyconnectedLayer Forward CUDA method returned an error");
	}

	if (hipDeviceSynchronize() != hipError_t::hipSuccess)
	{
		throw std::runtime_error("FullyconnectedLayer Forward CUDA syncronize returned an error");
	}
}

void ConvLayer_Backward(ConvNode *node, double* filters, double* backFilterCollation, double* backFilters, LayerSize filterSize, int filterCount, LayerSize layerSize, LayerSize previousLayerSize, LayerSize nextLayerSize, double *previousLayerOutput, double *nextLayerOutput, double *output, int pad, double learnRate)
{
	// TODO: I ASSUME THE PAD IS 1!!

	dim3 blocks(layerSize.width, layerSize.height, filterCount);
	ConvLayer_Backward_update_back_filters_cu <<<blocks, 1>>>(node, filters, backFilterCollation, backFilters, filterSize, filterCount, layerSize, previousLayerSize, nextLayerSize, previousLayerOutput, nextLayerOutput, output, pad, learnRate);
	//ConvLayer_Backward_cu_2 << <1, 1 >> >(node, filters, backFilters, filterSize, filterCount, layerSize, previousLayerSize, nextLayerSize, previousLayerOutput, nextLayerOutput, output, pad, learnRate);

	dim3 bfblocks(filterSize.width, filterSize.height, filterCount);
	ConvLayer_Backward_update_back_filters_collate<<<bfblocks , 1>>>(backFilterCollation, backFilters, filterSize, layerSize);


	//dim3 bfblocks(filterSize.width, filterSize.height, filterCount);
	//dim3 bthreads(layerSize.width, layerSize.height, 1);
//	ConvLayer_Backward_update_back_filters_cu <<<bfblocks, bthreads >>>(node, filters, backFilterCollation, backFilters, filterSize, filterCount, layerSize, previousLayerSize, nextLayerSize, previousLayerOutput, nextLayerOutput, output, pad, learnRate);
	
	dim3 bblocks(layerSize.width, layerSize.height, filterSize.depth);
	//dim3 bthreads2(filterSize.width, filterSize.height, 1);
	ConvLayer_Backward_update_output_cu <<<bblocks, 1 >>>(node, filters, backFilters, filterSize, filterCount, layerSize, previousLayerSize, nextLayerSize, previousLayerOutput, nextLayerOutput, output, pad, learnRate);


	//ConvLayer_Backward_cu_2 << <1, 1 >> >(node, filters, backFilters, filterSize, filterCount, layerSize, previousLayerSize, nextLayerSize, previousLayerOutput, nextLayerOutput, output, pad, learnRate);
	//ConvLayer_Update_Backward_filter_cu <<<filterCount, 1 >>>(filters, backFilters, filterSize, learnRate);

	if (hipGetLastError() != hipError_t::hipSuccess)
	{
		throw std::runtime_error("FullyconnectedLayer Forward CUDA method returned an error");
	}

	if (hipDeviceSynchronize() != hipError_t::hipSuccess)
	{
		throw std::runtime_error("FullyconnectedLayer Forward CUDA syncronize returned an error");
	}
}