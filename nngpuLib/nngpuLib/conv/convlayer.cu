#include "hip/hip_runtime.h"
#pragma once
#include <stdio.h>
#include <stdexcept>

#include "convlayer.h"
#include "layersize.h"
#include "hip/hip_runtime.h"
#include "math.h"
#include ""
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__ void ConvLayer_Forward_cu(ConvNode *node, double* filters, LayerSize filterSize, LayerSize layerSize, LayerSize previousLayerSize, double *previousLayerOutput, double *output, int pad)
{
	int posx = blockIdx.x - pad;
	int posy = blockIdx.y - pad;
	double val = 0;
	double* filter = filters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.z);

	for (int filterPosy = 0; filterPosy < filterSize.height;filterPosy++)
	{
		for (int filterPosx = 0; filterPosx < filterSize.width; filterPosx++)
		{
			if (filterPosy + posy >= 0 &&
				filterPosy + posy < previousLayerSize.height &&
				filterPosx + posx >= 0 &&
				filterPosx + posx < previousLayerSize.width)
			{
				for (int d = 0; d < filterSize.depth; d++)
				{
					int index1 = ((filterPosy * filterSize.width) + filterPosx) * filterSize.depth + d;
					int index2 = (((posy + filterPosy) * previousLayerSize.width) + posx + filterPosx) * previousLayerSize.depth + d;

					if (index1 > filterSize.width * filterSize.height * filterSize.depth)
					{
						val = 0;
					}

					if (index2 > previousLayerSize.width * previousLayerSize.height * previousLayerSize.depth)
					{
						val = 0;
					}
					val += filter[index1] * previousLayerOutput[index2];
					/*
					if (blockIdx.x == 1 && blockIdx.y == 0)
					{
						printf("gradient x: %i, y: %i, d2: %i, layerSize.width: %i !\n", blockIdx.x, blockIdx.y, blockIdx.z, layerSize.width);
						printf("filterPosx: %i, filterPosy: %i, posx: %i, posy: %i\n", filterPosx, filterPosy, posx, posy);
						printf("filter[index1]: %f, previousLayerOutput[index2]: %f\n", filter[index1], previousLayerOutput[index2]);
						printf("d: %i\n", d);
						printf("index1: %i\n index2: %i\n", index1, index2);
					}*/

				}
			}
		}
	}

	val += node[blockIdx.z].bias;

	output[((blockIdx.y * layerSize.width) + blockIdx.x) * layerSize.depth + blockIdx.z] = val;
	/*if (blockIdx.x == 1 && blockIdx.y == 0) {
		printf("output[1]: %f\n", output[1]);
	}*/
}

__global__ void ConvLayer_Backward_update_output_cu(ConvNode *node, double* filters, double* backFilters, LayerSize filterSize, int filterCount, LayerSize layerSize, LayerSize previousLayerSize, LayerSize nextLayerSize, double *previousLayerOutput, double *nextLayerOutput, double *output, int pad, double learnRate)
{
	// TODO: ASSUMING PAD OF 1!!

	//int posx = blockIdx.x - pad;
	//int posy = blockIdx.y - pad;
	int d = threadIdx.x;

	//unsigned int index1 = ((layerSize.width * (filterPosy + posy)) + filterPosx + posx) * previousLayerSize.depth + d;
	unsigned int index1 = ((layerSize.width * blockIdx.y) + blockIdx.x) * previousLayerSize.depth + d;
	//unsigned int filterStartX = filterSize.width - pad - 1 + blockIdx.x;
	//unsigned int filterStartY = filterSize.height - pad - 1 + blockIdx.y;

	int fpxStart = -pad;
	int filterStartPosx = filterSize.width - 1;
	if ((int)blockIdx.x - pad < 0)
	{
		fpxStart = 0;
		filterStartPosx = filterSize.width - pad - 1;
	}

	int fpyStart = -pad;
	int filterStartPosy = filterSize.height - 1;
	if ((int)blockIdx.y - pad < 0)
	{
		fpyStart = 0;
		filterStartPosy = filterSize.height - pad - 1;
	}

	int filterEndPosx = 0;
	if ((int)blockIdx.x + filterSize.width - pad > layerSize.width)
	{
		filterEndPosx = (int)blockIdx.x + filterSize.width - pad - layerSize.width;
	}

	int filterEndPosy = 0;
	if ((int)blockIdx.y + filterSize.height - pad > layerSize.height)
	{
		filterEndPosy = (int)blockIdx.y + filterSize.height - pad - layerSize.height;
	}
	/*
	if (blockIdx.x == 11 && blockIdx.y == 0)
	{
	printf("filterStartPosx: %i\n", filterStartPosx);
	printf("filterStartPosy: %i\n", filterStartPosy);

	printf("filterEndPosx: %i\n", filterEndPosx);
	printf("filterEndPosy: %i\n", filterEndPosy);

	printf("?? (int)blockIdx.x + filterSize.width: %i\n", (int)blockIdx.x + filterSize.width);
	printf("?? (int)blockIdx.y + filterSize.height: %i\n", (int)blockIdx.y + filterSize.height);
	printf("?? layerSize.height: %i\n", layerSize.height);
	}*/


	for (int filterIndex = 0; filterIndex < filterCount; filterIndex++)
	{
		double* filter = filters + (filterSize.width * filterSize.height * filterSize.depth * filterIndex);


		for (int fpy = fpyStart, int filterPosy = filterStartPosy; filterPosy >= filterEndPosy; fpy++, filterPosy--)
		{
			for (int fpx = fpxStart, int filterPosx = filterStartPosx; filterPosx >= filterEndPosx; fpx++, filterPosx--)
			{

				//int fpx = filterStartX + filterPosx - blockIdx.x + filterSize.width - 1;
				//int fpy = filterStartY + filterPosy - blockIdx.y + filterSize.height - 1;
				double gradient = nextLayerOutput[((layerSize.width * (blockIdx.y + fpy)) + (blockIdx.x + fpx)) * nextLayerSize.depth + filterIndex];
				/*
				if (blockIdx.x == 11 && blockIdx.y == 0)
				{
				printf("fpx: %i, fpy: %i\n", fpx, fpy);
				printf("gradient x: %i, y: %i nextLayerSize.depth: %i, d2: %i, layerSize.width: %i \n", blockIdx.x + fpx, blockIdx.y + fpy, nextLayerSize.depth, filterIndex, layerSize.width);
				}*/

				int index2 = ((filterSize.width * filterPosy) + filterPosx) * filterSize.depth + d;

				output[index1] += filter[index2] * gradient;
				/*
				if (blockIdx.x == 11 && blockIdx.y == 0)
				{
				printf("gradient: %f\n", gradient);
				printf("filter: %i\n", filterIndex);
				printf("index1: %i\n index2: %i\n", index1, index2);
				}*/

			}
		}
	}
	/*
	if (blockIdx.x == 11 && blockIdx.y == 0)
	{
	printf("index1 %i\n", index1);
	printf("output[index1] %f\n", output[index1]);
	}*/

	//node->bias += gradient * learnRate;
}

__global__ void ConvLayer_Backward_update_bias_cu(ConvNode *node, LayerSize layerSize, LayerSize nextLayerSize, double *nextLayerOutput, double learnRate)
{
	for (int y = 0; y < layerSize.height; y++)
	{
		for (int x = 0; x < layerSize.width; x++)
		{
			double gradient = nextLayerOutput[((layerSize.width * y) + x) * nextLayerSize.depth + blockIdx.x];

			node[blockIdx.x].bias += gradient * learnRate;
		}
	}
}

__global__ void ConvLayer_Backward_back_filters_cu2(double* backFilters, LayerSize filterSize, LayerSize layerSize, LayerSize previousLayerSize, double *previousLayerOutput, LayerSize nextLayerSize, double *nextLayerOutput, int pad, int* backFilterLookUp)
{
	// do each pixel in each filter!
	// 
	// blockIdx.x = filter x
	// blockIdx.y = filter y
	// blockIdx.z = filter index (count)
	// threadIdx.x = filterSize.depth

//	int ii = 0;

	//	int index2 = ((filterSize.width * blockIdx.x) + blockIdx.y) * filterSize.depth + blockIdx.z;
	//int d = 0;
	//for (int d = 0; d < filterSize.depth; d++)
	{
		double* backFilter = backFilters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.z);

		int index2 = ((filterSize.width * blockIdx.y) + blockIdx.x) * filterSize.depth;
		int* lookUp = backFilterLookUp + (((filterSize.width * blockIdx.y) + blockIdx.x) * layerSize.width * layerSize.height * 2);
		index2 += threadIdx.x;

		for (int y = 0; y < layerSize.height; y++)
		{
			for (int x = 0; x < layerSize.width; x++)
			{
				int index1 = *lookUp;
				lookUp++;
				int gradIndex = *lookUp;
				lookUp++;

				if (index1 >= 0)
				{
					gradIndex += blockIdx.z;

					//double gradient = nextLayerOutput[((layerSize.width * (y - pad)) + (x - pad)) * nextLayerSize.depth + blockIdx.z];
					double gradient = nextLayerOutput[gradIndex];

					//int index1 = ((layerSize.width * (y - pad)) + (x - pad)) * previousLayerSize.depth + threadIdx.x;

					//double gradient = nextLayerOutput[((layerSize.width * y) + x) * nextLayerSize.depth + d2];
					//int index1 = ((layerSize.width * (filterPosy + posy)) + filterPosx + posx) * previousLayerSize.depth + d;

					index1 += threadIdx.x;
					/*
					if (index2 == 32 && blockIdx.z == 0)
					{
						if (ii < 4000) {
							printf("iiiii: %i, index1: %i, index2: %i, filterPosx: %i, filterPosy: %i, posx: %i, posy: %i, gradient: %f, d: %i, d2: %i, x: %i, y: %i \n", ii, index1, index2, blockIdx.x, blockIdx.y, x, y, gradient, threadIdx.x, blockIdx.z, x - pad, y - pad);
							ii++;
						}
					}*/

					backFilter[index2] += previousLayerOutput[index1] * gradient;
				}
			}
		}
		/*
		if (index2 == 32 && blockIdx.z == 0)
		{
			printf("backFilters[32] %f\n", backFilters[32]);
		}*/
	}
}

__global__ void ConvLayer_Update_Backward_filter_cu(double* filters, double* backFilters, LayerSize filterSize, double learnRate)
{
	double* filter = filters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.x);
	double* backFilter = backFilters + (filterSize.width * filterSize.height * filterSize.depth * blockIdx.x);

	int size = filterSize.width * filterSize.height * filterSize.depth;
	for (int index = 0; index < size; index++)
	{
		filter[index] += backFilter[index] * learnRate;
	}
}

void ConvLayer_Forward(ConvNode *node, double* filters, LayerSize filterSize, int filterCount, LayerSize layerSize, LayerSize previousLayerSize, double *previousLayerOutput, double *output, int pad)
{
	dim3 blocks(layerSize.width, layerSize.height, filterCount);
	ConvLayer_Forward_cu <<<blocks, 1>>>(node, filters, filterSize, layerSize, previousLayerSize, previousLayerOutput, output, pad);

	LayerSynchronize();
}

void ConvLayer_Backward(ConvNode *node, double* filters, double* backFilters, LayerSize filterSize, int filterCount, LayerSize layerSize, LayerSize previousLayerSize, LayerSize nextLayerSize, double *previousLayerOutput, double *nextLayerOutput, double *output, int pad, double learnRate, int* backFilterLookUp, int backFilterLookUpSize)
{
	// TODO: I ASSUME THE PAD IS 1!!

	dim3 bffblocks(filterSize.width, filterSize.height, filterCount);
	ConvLayer_Backward_back_filters_cu2 <<<bffblocks, filterSize.depth>>>(backFilters, filterSize, layerSize, previousLayerSize, previousLayerOutput, nextLayerSize, nextLayerOutput, pad, backFilterLookUp);

	LayerSynchronize();

	dim3 bblocks(layerSize.width, layerSize.height, 1);
	ConvLayer_Backward_update_output_cu << <bblocks, filterSize.depth >> >(node, filters, backFilters, filterSize, filterCount, layerSize, previousLayerSize, nextLayerSize, previousLayerOutput, nextLayerOutput, output, pad, learnRate);

	LayerSynchronize();

	ConvLayer_Update_Backward_filter_cu <<<filterCount, 1 >>>(filters, backFilters, filterSize, learnRate);

	LayerSynchronize();

	ConvLayer_Backward_update_bias_cu<<<filterCount, 1>>>(node, layerSize, nextLayerSize, nextLayerOutput, learnRate);

	LayerSynchronize();
}